#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix_world.h>

using namespace optix;

// This is to be plugged into an RTgeometry object to represent
// a triangle mesh with a vertex buffer of triangle soup (triangle list)
// with an interleaved position, normal, texturecoordinate layout.

struct Vertex
{
  float3 position;
  float3 normal;
};

rtBuffer<Vertex> vertex_buffer;
rtBuffer<int3>   index_buffer;

rtBuffer<uint>      material_buffer; // per-face material index
rtDeclareVariable(float3, texcoord, attribute texcoord, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );


RT_PROGRAM void mesh_intersect( int primIdx )
{
  int3 v_idx = index_buffer[primIdx];

  float3 p0 = vertex_buffer[ v_idx.x ].position;
  float3 p1 = vertex_buffer[ v_idx.y ].position;
  float3 p2 = vertex_buffer[ v_idx.z ].position;

  // Intersect ray with triangle
  float3 n;
  float  t, beta, gamma;
  if( intersect_triangle( ray, p0, p1, p2, n, t, beta, gamma ) ) {

    if(  rtPotentialIntersection( t ) ) {

      float3 n0 = vertex_buffer[ v_idx.x ].normal;
      float3 n1 = vertex_buffer[ v_idx.y ].normal;
      float3 n2 = vertex_buffer[ v_idx.z ].normal;
      shading_normal   = normalize( n0*(1.0f-beta-gamma) + n1*beta + n2*gamma );
      geometric_normal = normalize( n );

      rtReportIntersection(material_buffer[primIdx]);
    }
  }
}


RT_PROGRAM void mesh_bounds (int primIdx, optix::Aabb* aabb)
{
  const int3 v_idx = index_buffer[primIdx];

  const float3 v0 = vertex_buffer[ v_idx.x ].position;
  const float3 v1 = vertex_buffer[ v_idx.y ].position;
  const float3 v2 = vertex_buffer[ v_idx.z ].position;
  const float  area = length(cross(v1-v0, v2-v0));

  if(area > 0.0f && !isinf(area)) {
    aabb->m_min = fminf( fminf( v0, v1), v2 );
    aabb->m_max = fmaxf( fmaxf( v0, v1), v2 );
  } else {
    aabb->invalidate();
  }
}

