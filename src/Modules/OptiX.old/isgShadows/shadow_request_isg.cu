#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtBuffer<float,  2>          shadow_buffer;
rtTextureSampler<float4, 2>  request_texture;

rtDeclareVariable(uint, shadow_ray_type, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

rtDeclareVariable(float3, light_pos, , );
rtDeclareVariable(rtObject, shadow_casters, , );

struct PerRayData_shadow
{
  float3 attenuation;
};

RT_PROGRAM void shadow_request()
{
  float3 ray_origin = make_float3(tex2D(request_texture, launch_index.x, launch_index.y));

  PerRayData_shadow prd;
  prd.attenuation = make_float3(-1);

  if( !isnan(ray_origin.x) ) {
    float3 L = light_pos-ray_origin;
    float dist = sqrtf(dot(L,L));
    float3 ray_direction = L/dist;
    optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, shadow_ray_type, scene_epsilon, dist);
    rtTrace(shadow_casters, ray, prd);
  }

  shadow_buffer[launch_index] = prd.attenuation.x;
}

RT_PROGRAM void exception()
{
}
