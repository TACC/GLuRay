#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtBuffer<float4, 2>                                      result_buffer;
rtTextureSampler<uchar4, 2, hipReadModeNormalizedFloat> input_texture;
// This line would also implicitly cast up to floats:
// rtTextureSampler<float4, 2> input_texture;
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

RT_PROGRAM void draw_texture()
{
  size_t2 screen = result_buffer.size();
  float2 uv = make_float2(launch_index) / make_float2(screen);
  
  result_buffer[launch_index] = tex2D(input_texture, uv.x, uv.y);
}
