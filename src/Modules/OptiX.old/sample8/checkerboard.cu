#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtDeclareVariable(rtObject,     top_object, , );
rtDeclareVariable(float,        scene_epsilon, , );
rtDeclareVariable(int,          max_depth, , );
rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, shadow_ray_type, , );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, isect_dist, rtIntersectionDistance, );

rtDeclareVariable(float3,       tile_size, , ); 
rtDeclareVariable(float3,       tile_color_dark, , );
rtDeclareVariable(float3,       tile_color_light, , );
rtDeclareVariable(float3,       light_direction, , );

struct PerRayData_radiance
{
  float3 result;
  float importance;
  int depth;
};

struct PerRayData_shadow
{
  float3 attenuation;
};

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

// -----------------------------------------------------------------------------

RT_PROGRAM void closest_hit_radiance()
{
  const float3 h = ray.origin + isect_dist * ray.direction;
  const float3 l = normalize(light_direction);
  const float3 n = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));

  const float3 p = h / tile_size;
  float3 result = (static_cast<int>( floorf(p.x) + floorf(p.y) + floorf(p.z) ) & 1) ?
                  tile_color_light : tile_color_dark;

  const float cos_theta = dot(l, n);
  if (cos_theta > 0.01f)
  {
    optix::Ray shadow_ray = optix::make_Ray( h, l, shadow_ray_type, scene_epsilon, RT_DEFAULT_MAX );
    PerRayData_shadow shadow_prd;
    shadow_prd.attenuation = make_float3(1.0f);
  
    rtTrace( top_object, shadow_ray, shadow_prd );
  
    result *= 0.25f + (cos_theta * shadow_prd.attenuation * 0.75f);
  }
  else
    result *= 0.25f;

  prd_radiance.result = result;
}

// -----------------------------------------------------------------------------

RT_PROGRAM void any_hit_shadow()
{
  // this material is opaque, so it fully attenuates all shadow rays
  prd_shadow.attenuation = make_float3(0.0f);
  rtTerminateRay();
}
