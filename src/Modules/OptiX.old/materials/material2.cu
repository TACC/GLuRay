#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include "materials.h"

rtDeclareVariable(float3, Ka, , );
rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(float3, ambient_light_color, , );
rtBuffer<BasicLight> lights;

rtDeclareVariable(float3, shading_normal,   attribute shading_normal, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );

// Semantic variables are maintained by the Optix engine
rtDeclareVariable(optix::Ray, ray, rtCurrentRay,);
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

//
// Lambertian shading with ambient light component 
// 

RT_PROGRAM void closest_hit_radiance()
{
  float3 world_geo_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 world_shade_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 ffnormal           = faceforward( world_shade_normal, -ray.direction, world_geo_normal );

  // Ambient component
  float3 ambient = Ka * ambient_light_color;

  // Lambertian lighting
  float3 hit_point = ray.origin + t_hit * ray.direction;

  float3 direct = make_float3( 0.0f );
  for(int i = 0; i < lights.size(); ++i) {
    BasicLight light = lights[i];
    float3 L  = normalize(light.pos - hit_point);
    float nDl = dot( ffnormal, L);
    if( nDl > 0.0f )
      direct += Kd * nDl * light.color;
  }
  prd_radiance.result = direct + ambient;
}


