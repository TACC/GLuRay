#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include "materials.h"

rtDeclareVariable(float3,   Ka, , );
rtDeclareVariable(float3,   Kd, , );
rtDeclareVariable(float3,   Ks, , );
rtDeclareVariable(float,    phong_exp, , );
rtDeclareVariable(float3,   ambient_light_color, , );
rtDeclareVariable(int,      max_depth, , );
rtDeclareVariable(float,    scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );
rtBuffer<BasicLight> lights;

rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );

// Semantic variables are maintained by the Optix engine
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

//
// Phong shading with reflections 
// 

RT_PROGRAM void closest_hit_radiance()
{
  float3 world_geo_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 world_shade_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 ffnormal           = faceforward( world_shade_normal, -ray.direction, world_geo_normal );

  // Ambient component
  float3 ambient = Ka * ambient_light_color;

  // Phong lighting
  float3 hit_point = ray.origin + t_hit * ray.direction;

  float3 direct = make_float3( 0.0f );
  for(int i = 0; i < lights.size(); ++i) {
    BasicLight light = lights[i];
    float3 L  = normalize(light.pos - hit_point);
    float nDl = dot( ffnormal, L);
    if( nDl > 0.0f ) {
      // Lambertian
      float3 diffuse = Kd * nDl * light.color;

      // Phong highlight
      float3 H = normalize(L - ray.direction);
      float nDh = dot( ffnormal, H );
      float3 specular = make_float3( 0.0f );
      if( nDh > 0.0f )
        specular = Ks * light.color * powf( nDh, phong_exp );

      direct += diffuse + specular;
    }
  }

  // reflection ray
  float3 reflected = make_float3( 0.1f );
  if( prd_radiance.depth < max_depth) {
    PerRayData_radiance refl_prd;
    refl_prd.depth = prd_radiance.depth+1;
    float3 R = reflect( ray.direction, ffnormal );
    optix::Ray refl_ray( hit_point, R, 0, scene_epsilon, RT_DEFAULT_MAX );
    rtTrace(top_object, refl_ray, refl_prd);
    reflected = Ks * refl_prd.result;
  }

  prd_radiance.result = ambient + direct + reflected;
}


