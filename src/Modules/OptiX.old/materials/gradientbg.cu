#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtDeclareVariable(float3, background_light, , ); // horizon color
rtDeclareVariable(float3, background_dark, , );  // zenith color
rtDeclareVariable(float3, up, , );               // global up vector

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

struct PerRayData_radiance
{
  float3 result;
  float  importance;
  int    depth;
};

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

// -----------------------------------------------------------------------------

RT_PROGRAM void miss()
{
  const float t = max(dot(ray.direction, up), 0.0f);
  const float3 result = lerp(background_light, background_dark, t);

  prd_radiance.result = result;
}
