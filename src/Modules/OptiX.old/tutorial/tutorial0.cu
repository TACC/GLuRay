#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include "tutorial.h"

rtDeclareVariable(float3, shading_normal,   attribute shading_normal, ); 

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

rtDeclareVariable(optix::Ray, ray,          rtCurrentRay, );
rtDeclareVariable(uint2,      launch_index, rtLaunchIndex, );

rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(float,        scene_epsilon, , );
rtDeclareVariable(rtObject,     top_object, , );


//
// Pinhole camera implementation
//
rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtBuffer<uchar4, 2>              output_buffer;

RT_PROGRAM void pinhole_camera()
{
  size_t2 screen = output_buffer.size();

  float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f;
  float3 ray_origin = eye;
  float3 ray_direction = normalize(d.x*U + d.y*V + W);

  optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon );

  PerRayData_radiance prd;
  prd.importance = 1.f;
  prd.depth = 0;

  rtTrace(top_object, ray, prd);

  output_buffer[launch_index] = make_color( prd.result );
}

//
// Returns solid color for miss rays
//
rtDeclareVariable(float3, bg_color, , );
RT_PROGRAM void miss()
{
  prd_radiance.result = bg_color;
}
  

//
// Returns shading normal as the surface shading result
// 
RT_PROGRAM void closest_hit_radiance0()
{
  prd_radiance.result = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal))*0.5f + 0.5f;
}
 

//
// Set pixel to solid color upon failur
//
RT_PROGRAM void exception()
{
  output_buffer[launch_index] = make_color( bad_color );
}
