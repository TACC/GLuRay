#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include "tutorial.h"

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, ); 

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

rtDeclareVariable(optix::Ray, ray,          rtCurrentRay, );
rtDeclareVariable(float,      t_hit,        rtIntersectionDistance, );
rtDeclareVariable(uint2,      launch_index, rtLaunchIndex, );

rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(float,        scene_epsilon, , );
rtDeclareVariable(rtObject,     top_object, , );


//
// Pinhole camera implementation
//
rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtBuffer<uchar4, 2>              output_buffer;

RT_PROGRAM void pinhole_camera()
{
  size_t2 screen = output_buffer.size();

  float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f;
  float3 ray_origin = eye;
  float3 ray_direction = normalize(d.x*U + d.y*V + W);

  optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon );

  PerRayData_radiance prd;
  prd.importance = 1.f;
  prd.depth = 0;

  rtTrace(top_object, ray, prd);

  output_buffer[launch_index] = make_color( prd.result );
}


//
// Returns solid color for miss rays
//
rtDeclareVariable(float3, bg_color, , );
RT_PROGRAM void miss()
{
  prd_radiance.result = bg_color;
}
  

//
// (UPDATED)
// Implements basic phong shading model -- lambertian plus highlights 
//
rtDeclareVariable(float3,  Ka, , );
rtDeclareVariable(float3,  Ks, , );
rtDeclareVariable(float,   phong_exp, , );
rtDeclareVariable(float3,  Kd, , );
rtDeclareVariable(float3,  ambient_light_color, , );
rtBuffer<BasicLight>       lights;

RT_PROGRAM void closest_hit_radiance2()
{
  float3 world_geo_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 world_shade_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 ffnormal     = faceforward( world_shade_normal, -ray.direction, world_geo_normal );
  float3 color = Ka * ambient_light_color;

  float3 hit_point = ray.origin + t_hit * ray.direction;

  for(int i = 0; i < lights.size(); ++i) {
    BasicLight light = lights[i];
    float3 L = normalize(light.pos - hit_point);
    float nDl = dot( ffnormal, L);

    if( nDl > 0 ){
      float3 Lc = light.color;
      color += Kd * nDl * Lc;

      float3 H = normalize(L - ray.direction);
      float nDh = dot( ffnormal, H );
      if(nDh > 0)
        color += Ks * Lc * pow(nDh, phong_exp);

    }
  }
  prd_radiance.result = color;
}
  

//
// Set pixel to solid color upon failure
//
RT_PROGRAM void exception()
{
  output_buffer[launch_index] = make_color( bad_color );
}
