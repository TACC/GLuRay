#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix;

rtDeclareVariable(float3, boxmin, , );
rtDeclareVariable(float3, boxmax, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, texcoord, attribute texcoord, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 

static __device__ float3 boxnormal(float t)
{
  float3 t0 = (boxmin - ray.origin)/ray.direction;
  float3 t1 = (boxmax - ray.origin)/ray.direction;
  float3 neg = make_float3(t==t0.x?1:0, t==t0.y?1:0, t==t0.z?1:0);
  float3 pos = make_float3(t==t1.x?1:0, t==t1.y?1:0, t==t1.z?1:0);
  return pos-neg;
}

RT_PROGRAM void box_intersect(int)
{
  float3 t0 = (boxmin - ray.origin)/ray.direction;
  float3 t1 = (boxmax - ray.origin)/ray.direction;
  float3 near = fminf(t0, t1);
  float3 far = fmaxf(t0, t1);
  float tmin = fmaxf( near );
  float tmax = fminf( far );

  if(tmin <= tmax) {
    bool check_second = true;
    if( rtPotentialIntersection( tmin ) ) {
       texcoord = make_float3( 0.0f );
       shading_normal = geometric_normal = boxnormal( tmin );
       if(rtReportIntersection(0))
         check_second = false;
    } 
    if(check_second) {
      if( rtPotentialIntersection( tmax ) ) {
        texcoord = make_float3( 0.0f );
        shading_normal = geometric_normal = boxnormal( tmax );
        rtReportIntersection(0);
      }
    }
  }
}

RT_PROGRAM void box_bounds (int, float result[6])
{
  optix::Aabb* aabb = (optix::Aabb*)result;
  aabb->set(boxmin, boxmax);
}
