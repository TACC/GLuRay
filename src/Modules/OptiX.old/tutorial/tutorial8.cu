#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include "tutorial.h"

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, ); 

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );

rtDeclareVariable(optix::Ray, ray,          rtCurrentRay, );
rtDeclareVariable(float,      t_hit,        rtIntersectionDistance, );
rtDeclareVariable(uint2,      launch_index, rtLaunchIndex, );

rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, shadow_ray_type , , );
rtDeclareVariable(float,        scene_epsilon, , );
rtDeclareVariable(rtObject,     top_object, , );


//
// Pinhole camera implementation
//
rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtBuffer<uchar4, 2>              output_buffer;

RT_PROGRAM void pinhole_camera()
{
  size_t2 screen = output_buffer.size();

  float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f;
  float3 ray_origin = eye;
  float3 ray_direction = normalize(d.x*U + d.y*V + W);

  optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon );

  PerRayData_radiance prd;
  prd.importance = 1.f;
  prd.depth = 0;

  rtTrace(top_object, ray, prd);

  output_buffer[launch_index] = make_color( prd.result );
}


//
// Environment map background
//
rtTextureSampler<float4, 2> envmap;
RT_PROGRAM void envmap_miss()
{
  float theta = atan2f( ray.direction.x, ray.direction.z );
  float phi   = M_PIf * 0.5f -  acosf( ray.direction.y );
  float u     = (theta + M_PIf) * (0.5f * M_1_PIf);
  float v     = 0.5f * ( 1.0f + sin(phi) );
  prd_radiance.result = make_float3( tex2D(envmap, u, v) );
}


//
// Terminates and fully attenuates ray after any hit
//
RT_PROGRAM void any_hit_shadow()
{
  // this material is opaque, so it fully attenuates all shadow rays
  prd_shadow.attenuation = make_float3(0);

  rtTerminateRay();
}
  

//
// (NEW)
// Procedural rusted metal surface shader
//

/*
 * Translated to CUDA C from Larry Gritz's LGRustyMetal.sl shader found at:
 * http://renderman.org/RMR/Shaders/LGShaders/LGRustyMetal.sl
 *
 * Used with permission from tal AT renderman DOT org.
 */

rtDeclareVariable(float,   metalKa, , ) = 1;
rtDeclareVariable(float,   metalKs, , ) = 1;
rtDeclareVariable(float,   metalroughness, , ) = .1;
rtDeclareVariable(float,   rustKa, , ) = 1;
rtDeclareVariable(float,   rustKd, , ) = 1;
rtDeclareVariable(float3,  rustcolor, , ) = {.437, .084, 0};
rtDeclareVariable(float3,  metalcolor, , ) = {.7, .7, .7};
rtDeclareVariable(float,   txtscale, , ) = .02;
rtDeclareVariable(float,   rusty, , ) = 0.2;
rtDeclareVariable(float,   rustbump, , ) = 0.85;
rtDeclareVariable(float3,  ambient_light_color, , );
rtBuffer<BasicLight>       lights;   
rtDeclareVariable(rtObject, top_shadower, , );
rtDeclareVariable(float,   importance_cutoff, , );
rtDeclareVariable(int,     max_depth, , );
rtDeclareVariable(float3,  reflectivity_n, , );
#define MAXOCTAVES 6

rtTextureSampler<float, 3> noise_texture;
static __device__ __inline__ float snoise(float3 p)
{
  return tex3D(noise_texture, p.x, p.y, p.z) * 2 -1;
}


RT_PROGRAM void box_closest_hit_radiance()
{
  float3 world_geo_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 world_shade_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 ffnormal     = faceforward( world_shade_normal, -ray.direction, world_geo_normal );
  float3 hit_point = ray.origin + t_hit * ray.direction;

  /* Sum several octaves of abs(snoise), i.e. turbulence.  Limit the
   * number of octaves by the estimated change in PP between adjacent
   * shading samples.
   */
  float3 PP = txtscale * hit_point;
  float a = 1;
  float sum = 0;
  for(int i = 0; i < MAXOCTAVES; i++ ){
    sum += a * fabs(snoise(PP));
    PP *= 2.0f;
    a *= 0.5f;
  }

  /* Scale the rust appropriately, modulate it by another noise 
   * computation, then sharpen it by squaring its value.
   */
  float rustiness = step (1-rusty, clamp (sum,0.0f,1.0f));
  rustiness *= clamp (abs(snoise(PP)), 0.0f, .08f) / 0.08f;
  rustiness *= rustiness;

  /* If we have any rust, calculate the color of the rust, taking into
   * account the perturbed normal and shading like matte.
   */
  float3 Nrust = ffnormal;
  if (rustiness > 0) {
    /* If it's rusty, also add a high frequency bumpiness to the normal */
    Nrust = normalize(ffnormal + rustbump * snoise(PP));
    Nrust = faceforward (Nrust, -ray.direction, world_geo_normal);
  }

  float3 color = mix(metalcolor * metalKa, rustcolor * rustKa, rustiness) * ambient_light_color;
  for(int i = 0; i < lights.size(); ++i) {
    BasicLight light = lights[i];
    float3 L = normalize(light.pos - hit_point);
    float nmDl = dot( ffnormal, L);
    float nrDl = dot( Nrust, L);

    if( nmDl > 0.0f || nrDl > 0.0f ){
      // cast shadow ray
      PerRayData_shadow shadow_prd;
      shadow_prd.attenuation = make_float3(1.0f);
      float Ldist = length(light.pos - hit_point);
      optix::Ray shadow_ray( hit_point, L, shadow_ray_type, scene_epsilon, Ldist );
      rtTrace(top_shadower, shadow_ray, shadow_prd);
      float3 light_attenuation = shadow_prd.attenuation;

      if( fmaxf(light_attenuation) > 0.0f ){
        float3 Lc = light.color * light_attenuation;
        nrDl = max(nrDl * rustiness, 0.0f);
        color += rustKd * rustcolor * nrDl * Lc;

        float r = nmDl * (1.0f-rustiness);
        if(nmDl > 0.0f){
          float3 H = normalize(L - ray.direction);
          float nmDh = dot( ffnormal, H );
          if(nmDh > 0)
            color += r * metalKs * Lc * pow(nmDh, 1.f/metalroughness);
        }
      }

    }
  }

  float3 r = schlick(-dot(ffnormal, ray.direction), reflectivity_n * (1-rustiness));
  float importance = prd_radiance.importance * optix::luminance( r );

  // reflection ray
  if( importance > importance_cutoff && prd_radiance.depth < max_depth) {
    PerRayData_radiance refl_prd;
    refl_prd.importance = importance;
    refl_prd.depth = prd_radiance.depth+1;
    float3 R = reflect( ray.direction, ffnormal );
    optix::Ray refl_ray( hit_point, R, radiance_ray_type, scene_epsilon );
    rtTrace(top_object, refl_ray, refl_prd);
    color += r * refl_prd.result;
  }

  prd_radiance.result = color;
}
  

//
// Phong surface shading with shadows and schlick-approximated fresnel reflections.
// Uses procedural texture to determine diffuse response.
//
rtDeclareVariable(float3,   Ka, , );
rtDeclareVariable(float3,   Ks, , );
rtDeclareVariable(float3,   Kd, , );
rtDeclareVariable(float,    phong_exp, , );
rtDeclareVariable(float3,   reflectivity, , );
rtDeclareVariable(float3,   tile_v0, , );
rtDeclareVariable(float3,   tile_v1, , );
rtDeclareVariable(float3,   crack_color, , );
rtDeclareVariable(float,    crack_width, , );

RT_PROGRAM void floor_closest_hit_radiance()
{
  float3 world_geo_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 world_shade_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 ffnormal     = faceforward( world_shade_normal, -ray.direction, world_geo_normal );
  float3 color = Ka * ambient_light_color;

  float3 hit_point = ray.origin + t_hit * ray.direction;

  float v0 = dot(tile_v0, hit_point);
  float v1 = dot(tile_v1, hit_point);
  v0 = v0 - floor(v0);
  v1 = v1 - floor(v1);

  float3 local_Kd;
  if( v0 > crack_width && v1 > crack_width ){
    local_Kd = Kd;
  } else {
    local_Kd = crack_color;
  }

  for(int i = 0; i < lights.size(); ++i) {
    BasicLight light = lights[i];
    float3 L = normalize(light.pos - hit_point);
    float nDl = dot( ffnormal, L);

    if( nDl > 0.0f ){
      // cast shadow ray
      PerRayData_shadow shadow_prd;
      shadow_prd.attenuation = make_float3(1.0f);
      float Ldist = length(light.pos - hit_point);
      optix::Ray shadow_ray( hit_point, L, shadow_ray_type, scene_epsilon, Ldist );
      rtTrace(top_shadower, shadow_ray, shadow_prd);
      float3 light_attenuation = shadow_prd.attenuation;

      if( fmaxf(light_attenuation) > 0.0f ){
        float3 Lc = light.color * light_attenuation;
        color += local_Kd * nDl * Lc;

        float3 H = normalize(L - ray.direction);
        float nDh = dot( ffnormal, H );
        if(nDh > 0)
          color += Ks * Lc * pow(nDh, phong_exp);
      }

    }
  }

  float3 r = schlick(-dot(ffnormal, ray.direction), reflectivity_n);
  float importance = prd_radiance.importance * optix::luminance( r );

  // reflection ray
  if( importance > importance_cutoff && prd_radiance.depth < max_depth) {
    PerRayData_radiance refl_prd;
    refl_prd.importance = importance;
    refl_prd.depth = prd_radiance.depth+1;
    float3 R = reflect( ray.direction, ffnormal );
    optix::Ray refl_ray( hit_point, R, radiance_ray_type, scene_epsilon );
    rtTrace(top_object, refl_ray, refl_prd);
    color += r * refl_prd.result;
  }

  prd_radiance.result = color;
}
  

//
// Set pixel to solid color upon failure
//
RT_PROGRAM void exception()
{
  output_buffer[launch_index] = make_color( bad_color );
}
