#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix;

// This is to be plugged into an RTgeometry object to represent a triangle mesh
// with a shared vertex buffer, triangle index buffer, and single material

rtBuffer<float3> vertex_buffer;     
rtBuffer<int3>   vindex_buffer;    // position indices 

rtDeclareVariable(float3, texcoord, attribute texcoord, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

RT_PROGRAM void mesh_intersect( int primIdx )
{
  int3 v_idx = vindex_buffer[primIdx];

  float3 p0 = vertex_buffer[ v_idx.x ];
  float3 p1 = vertex_buffer[ v_idx.y ];
  float3 p2 = vertex_buffer[ v_idx.z ];

  // Intersect ray with triangle
  float3 n;
  float  t, beta, gamma;
  if( intersect_triangle( ray, p0, p1, p2, n, t, beta, gamma ) ) {

    if(  rtPotentialIntersection( t ) ) {

      geometric_normal = normalize( n );
      shading_normal   = geometric_normal;
      texcoord = make_float3( 0.0f, 0.0f, 0.0f );

      rtReportIntersection( 0 );
    }
  }
}


RT_PROGRAM void mesh_bounds (int primIdx, float result[6])
{
  int3 v_idx = vindex_buffer[primIdx];

  float3 v0   = vertex_buffer[ v_idx.x ];
  float3 v1   = vertex_buffer[ v_idx.y ];
  float3 v2   = vertex_buffer[ v_idx.z ];
  const float  area = length(cross(v1-v0, v2-v0));

  optix::Aabb* aabb = (optix::Aabb*)result;

  if(area > 0.0f && !isinf(area)) {
    aabb->m_min = fminf( fminf( v0, v1), v2 );
    aabb->m_max = fmaxf( fmaxf( v0, v1), v2 );
  } else {
    aabb->invalidate();
  }
}

