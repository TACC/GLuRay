#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "phong.h"

using namespace optix;

rtTextureSampler<float4, 2>   ambient_map;         // 
rtTextureSampler<float4, 2>   diffuse_map;         // Correspond to OBJ mtl params
rtTextureSampler<float4, 2>   specular_map;        // 
rtDeclareVariable(float,      phong_exp, , );          //
rtDeclareVariable(int,        illum, , );              //

rtDeclareVariable(float3, texcoord, attribute texcoord, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 

RT_PROGRAM void any_hit_shadow()
{
  phongShadowed();
}

RT_PROGRAM void closest_hit_radiance()
{
  float3 direction              = ray.direction;
  float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 ffnormal               = faceforward( world_shading_normal, -direction, world_geometric_normal );
  float3 uv                     = texcoord;

  float3 black = make_float3(0.0f, 0.0f, 0.0f);

  // grab values from textures
  // support only MTL illumination modes 0-3 (Ks is for now used as reflectivity)
  // ( see http://local.wasp.uwa.edu.au/~pbourke/dataformats/mtl/ )
  float3 Kd =                       make_float3( tex2D( diffuse_map,  uv.x, uv.y ) );
  float3 Ka = (illum < 1) ? black : make_float3( tex2D( ambient_map,  uv.x, uv.y ) );
  float3 Ks = (illum < 2) ? black : make_float3( tex2D( specular_map, uv.x, uv.y ) );
  float3 Kr = (illum < 3) ? black : Ks;

  phongShade( Kd, Ka, Ks, ffnormal, phong_exp, Kr );
}
