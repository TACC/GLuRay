#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "helpers.h"
#include "commonStructs.h"

using namespace optix;

// global parameters
rtDeclareVariable(rtObject,     top_object, , );
rtDeclareVariable(rtObject,     top_shadower, , );
rtDeclareVariable(float,        scene_epsilon, , );
rtDeclareVariable(int,          max_depth, , );
rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, shadow_ray_type, , );
rtDeclareVariable(float3,       ambient_light_color, , );
rtDeclareVariable(float3,       jitter, , );

// surface parameters
rtDeclareVariable(float,        importance_cutoff, , );
rtDeclareVariable(float3,       cutoff_color, , );
rtDeclareVariable(float,        fresnel_exponent, , );
rtDeclareVariable(float,        fresnel_minimum, , );
rtDeclareVariable(float,        fresnel_maximum, , );
rtDeclareVariable(float3,       reflection_color, , );
rtDeclareVariable(int,          reflection_maxdepth, , );
rtDeclareVariable(float3,       Kd, , );
rtDeclareVariable(float3,       Ka, , );
rtDeclareVariable(float3,       Ks, , );
rtDeclareVariable(float,        exponent, , );

// surface textures
rtTextureSampler<float4, 2>     kd_map;

// hitpoint parameters
rtDeclareVariable(float3, texcoord, attribute texcoord, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 

// ray parameters
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, isect_dist, rtIntersectionDistance, );

struct PerRayData_radiance
{
  float3 result;
  float importance;
  int depth;
};

struct PerRayData_shadow
{
  float3 attenuation;
};

rtBuffer<BasicLight> lights;

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );

// -----------------------------------------------------------------------------

static __device__ __inline__ float3 TraceRay(float3 origin, float3 direction, int depth, float importance )
{
  optix::Ray ray = optix::make_Ray( origin, direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX );
  PerRayData_radiance prd;
  prd.depth = depth;
  prd.importance = importance;

  rtTrace( top_object, ray, prd );
  return prd.result;
}

// -----------------------------------------------------------------------------

RT_PROGRAM void closest_hit_radiance()
{
  // intersection vectors
  const float3 hit = ray.origin + isect_dist * ray.direction;            // hitpoint
  const float3 N   = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal)); // normal
  const float3 I   = ray.direction;                                            // incident direction
        float3 R   = reflect(I, N);                                            // reflection direction

  float depth = prd_radiance.depth;  

  float reflection = fresnel_schlick(-dot(N, I), fresnel_exponent, fresnel_minimum, fresnel_maximum);

  // we need not clamp this subtraction because after fresnel_schlick,
  // reflection is guaranteed to be <= fresnel_maximum
  float oneMinusFresnel = fresnel_maximum - reflection;

  // ambient
  float3 kd = Kd * make_float3( tex2D( kd_map, texcoord.x, texcoord.y ) );
  float3 result = oneMinusFresnel * Ka * ambient_light_color * kd;

  // direct lighting
  for (unsigned i=0; i<lights.size(); ++i)
  {
    float3 L = normalize(lights[i].pos - hit);

    // diffuse
    float3 diffuse = 1.0f/lights.size() * ( max(dot(N, L), 0.0f) * lights[i].color );
    result += oneMinusFresnel * diffuse * kd;

    // specular
    result += powf(max(dot(R, L), 0.0f) , exponent) * Ks;
  }

  // reflection
  if (depth < min(reflection_maxdepth, max_depth))
  {
    // phong lobe jittering
    float3 U, V, W;
    create_onb(R, U, V, W);
    R = sample_phong_lobe(make_float2(jitter.x, jitter.y), 4096.0f, U, V, W);

    // avoid directions below surface
    if (dot(R, N) < 0.01f)
      R = W;

    // shoot reflection ray
    float3 ntsc_luminance = {0.30f, 0.59f, 0.11f}; 
    float importance = prd_radiance.importance * reflection * dot( reflection_color, ntsc_luminance );
    float3 color = cutoff_color;
    if ( importance > importance_cutoff ) {
      color = TraceRay( hit, R, depth+1, importance );
    }
    result += reflection * reflection_color * color;
  }

  prd_radiance.result = result;
}

// -----------------------------------------------------------------------------

RT_PROGRAM void any_hit_shadow()
{
  prd_shadow.attenuation = make_float3(0.0f);
  rtTerminateRay();
}
