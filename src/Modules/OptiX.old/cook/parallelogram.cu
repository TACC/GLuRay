#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix_world.h>

using namespace optix;

rtDeclareVariable(float4, plane, , );
rtDeclareVariable(float3, v1, , );
rtDeclareVariable(float3, v2, , );
rtDeclareVariable(float3, anchor, , );
rtDeclareVariable(int, lgt_instance, , ) = {0};

rtDeclareVariable(float3, texcoord, attribute texcoord, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(int, lgt_idx, attribute lgt_idx, ); 
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

RT_PROGRAM void intersect(int primIdx)
{
  float3 n = make_float3( plane );
  float dt = dot(ray.direction, n );
  float t = (plane.w - dot(n, ray.origin))/dt;
  if( t > ray.tmin && t < ray.tmax ) {
    float3 p = ray.origin + ray.direction * t;
    float3 vi = p - anchor;
    float a1 = dot(v1, vi);
    if(a1 >= 0 && a1 <= 1){
      float a2 = dot(v2, vi);
      if(a2 >= 0 && a2 <= 1){
        if( rtPotentialIntersection( t ) ) {
          shading_normal = geometric_normal = n;
          texcoord = make_float3(a1,a2,0);
          lgt_idx = lgt_instance;
          rtReportIntersection( 0 );
        }
      }
    }
  }
}

RT_PROGRAM void bounds (int, float result[6])
{
  // v1 and v2 are scaled by 1./length^2.  Rescale back to normal for the bounds computation.
  const float3 tv1  = v1 / dot( v1, v1 );
  const float3 tv2  = v2 / dot( v2, v2 );
  const float3 p00  = anchor;
  const float3 p01  = anchor + tv1;
  const float3 p10  = anchor + tv2;
  const float3 p11  = anchor + tv1 + tv2;
  const float  area = length(cross(tv1, tv2));
  
  optix::Aabb* aabb = (optix::Aabb*)result;
  
  if(area > 0.0f && !isinf(area)) {
    aabb->m_min = fminf( fminf( p00, p01 ), fminf( p10, p11 ) );
    aabb->m_max = fmaxf( fmaxf( p00, p01 ), fmaxf( p10, p11 ) );
  } else {
    aabb->invalidate();
  }
}

