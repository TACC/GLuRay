#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

#include "helpers.h"

using namespace optix;

rtDeclareVariable(float4, sphere, , );
rtDeclareVariable(float3, rotation, , );

rtDeclareVariable(float3, matrix_row_0, , );
rtDeclareVariable(float3, matrix_row_1, , );
rtDeclareVariable(float3, matrix_row_2, , );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

rtDeclareVariable(float3, texcoord, attribute texcoord, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 

RT_PROGRAM void intersect(int primIdx)
{
  float3 center = make_float3(sphere);
  float3 O = ray.origin - center;
  float3 D = ray.direction;
  float radius = sphere.w;

  float b = dot(O, D);
  float c = dot(O, O)-radius*radius;
  float disc = b*b-c;
  if(disc > 0.0f){
    float sdisc = sqrtf(disc);
    float root1 = (-b - sdisc);
    bool check_second = true;
    if( rtPotentialIntersection( root1 ) ) {
      shading_normal = geometric_normal = (O + root1*D)/radius;

      float3 polar;
      polar.x = dot(matrix_row_0, geometric_normal);
      polar.y = dot(matrix_row_1, geometric_normal);
      polar.z = dot(matrix_row_2, geometric_normal);
      polar = optix::cart_to_pol(polar);

      texcoord = make_float3( polar.x*0.5f*M_1_PIf, (polar.y+M_PI_2f)*M_1_PIf, polar.z/radius );

      if(rtReportIntersection(0))
        check_second = false;
    } 
    if(check_second) {
      float root2 = (-b + sdisc);
      if( rtPotentialIntersection( root2 ) ) {
        shading_normal = geometric_normal = (O + root2*D)/radius;

        float3 polar;
        polar.x = dot(matrix_row_0, geometric_normal);
        polar.y = dot(matrix_row_1, geometric_normal);
        polar.z = dot(matrix_row_2, geometric_normal);
        polar = optix::cart_to_pol(polar);

        texcoord = make_float3( polar.x*0.5f*M_1_PIf, (polar.y+M_PI_2f)*M_1_PIf, polar.z/radius );

        rtReportIntersection(0);
      }
    }
  }
}

RT_PROGRAM void bounds (int, optix::Aabb* aabb)
{
  const float3 cen = make_float3( sphere );
  const float3 rad = make_float3( sphere.w );
  
  if( rad.x > 0.0f && !isinf(rad.x) ) {
    aabb->m_min = cen - rad;
    aabb->m_max = cen + rad;
  } else {
    aabb->invalidate();
  }
}

