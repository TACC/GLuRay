#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"
#include "path_tracer.h"
#include "random.h"

using namespace optix;

struct PerRayData_pathtrace
{
  float3 result;
  float3 radiance;
  float3 attenuation;
  float3 origin;
  float3 direction;
  unsigned int seed;
  int depth;
  int countEmitted;
  int done;
  int inside;
};

struct PerRayData_pathtrace_shadow
{
  bool inShadow;
};

// Scene wide
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(rtObject,      top_object, , );

// For camera
rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(unsigned int,  frame_number, , );
rtDeclareVariable(unsigned int,  sqrt_num_samples, , );
rtBuffer<float4, 2>              output_buffer;
rtBuffer<ParallelogramLight>     lights;

rtDeclareVariable(unsigned int,  pathtrace_ray_type, , );
rtDeclareVariable(unsigned int,  pathtrace_shadow_ray_type, , );
rtDeclareVariable(unsigned int,  rr_begin_depth, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, ); 

rtDeclareVariable(PerRayData_pathtrace, current_prd, rtPayload, );

rtDeclareVariable(optix::Ray, ray,          rtCurrentRay, );
rtDeclareVariable(float,      t_hit,        rtIntersectionDistance, );
rtDeclareVariable(uint2,      launch_index, rtLaunchIndex, );

static __device__ inline float3 powf(float3 a, float exp)
{
  return make_float3(powf(a.x, exp), powf(a.y, exp), powf(a.z, exp));
}

// For miss program
rtDeclareVariable(float3,       bg_color, , );

//-----------------------------------------------------------------------------
//
//  Camera program -- main ray tracing loop
//
//-----------------------------------------------------------------------------

RT_PROGRAM void pathtrace_camera()
{
  size_t2 screen = output_buffer.size();

  float2 inv_screen = 1.0f/make_float2(screen) * 2.f;
  float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;

  float2 jitter_scale = inv_screen / sqrt_num_samples;
  unsigned int samples_per_pixel = sqrt_num_samples*sqrt_num_samples;
  float3 result = make_float3(0.0f);

  unsigned int seed = tea<16>(screen.x*launch_index.y+launch_index.x, frame_number);
  do {
    unsigned int x = samples_per_pixel%sqrt_num_samples;
    unsigned int y = samples_per_pixel/sqrt_num_samples;
    float2 jitter = make_float2(x-rnd(seed), y-rnd(seed));
    float2 d = pixel + jitter*jitter_scale;
    float3 ray_origin = eye;
    float3 ray_direction = normalize(d.x*U + d.y*V + W);

    PerRayData_pathtrace prd;
    prd.result = make_float3(0.f);
    prd.attenuation = make_float3(1.f);
    prd.countEmitted = true;
    prd.done = false;
    prd.inside = false;
    prd.seed = seed;
    prd.depth = 0;

    for(;;) {
      Ray ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
      rtTrace(top_object, ray, prd);
      if(prd.done) {
        prd.result += prd.radiance * prd.attenuation;
        break;
      }

      // RR
      if(prd.depth >= rr_begin_depth){
        float pcont = fmaxf(prd.attenuation);
        if(rnd(prd.seed) >= pcont)
          break;
        prd.attenuation /= pcont;
      }
      prd.depth++;
      prd.result += prd.radiance * prd.attenuation;
      ray_origin = prd.origin;
      ray_direction = prd.direction;
    } // eye ray

    result += prd.result;
    seed = prd.seed;
  } while (--samples_per_pixel);

  float3 pixel_color = result/(sqrt_num_samples*sqrt_num_samples);

  if (frame_number > 1)
  {
    float a = 1.0f / (float)frame_number;
    float b = ((float)frame_number - 1.0f) * a;
    float3 old_color = make_float3(output_buffer[launch_index]);
    output_buffer[launch_index] = make_float4(a * pixel_color + b * old_color, 0.0f);
  }
  else
  {
    output_buffer[launch_index] = make_float4(pixel_color, 0.0f);
  }
}

rtDeclareVariable(float3,        emission_color, , );

RT_PROGRAM void diffuseEmitter()
{
  current_prd.radiance = current_prd.countEmitted? emission_color : make_float3(0.f);
  current_prd.done = true;
}

rtDeclareVariable(float3,        diffuse_color, , );

RT_PROGRAM void diffuse()
{
  float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

  float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

  float3 hitpoint = ray.origin + t_hit * ray.direction;
  current_prd.origin = hitpoint;

  float z1=rnd(current_prd.seed);
  float z2=rnd(current_prd.seed);
  float3 p;
  cosine_sample_hemisphere(z1, z2, p);
  float3 v1, v2;
  createONB(ffnormal, v1, v2);
  current_prd.direction = v1 * p.x + v2 * p.y + ffnormal * p.z;
  float3 normal_color = (normalize(world_shading_normal)*0.5f + 0.5f)*0.9;
  current_prd.attenuation = current_prd.attenuation * diffuse_color; // use the diffuse_color as the diffuse response
  current_prd.countEmitted = false;

  // Compute direct light...
  // Or shoot one...
  unsigned int num_lights = lights.size();
  float3 result = make_float3(0.0f);

  for(int i = 0; i < num_lights; ++i) {
    ParallelogramLight light = lights[i];
    float z1 = rnd(current_prd.seed);
    float z2 = rnd(current_prd.seed);
    float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

    float Ldist = length(light_pos - hitpoint);
    float3 L = normalize(light_pos - hitpoint);
    float nDl = dot( ffnormal, L );
    float LnDl = dot( light.normal, L );
    float A = length(cross(light.v1, light.v2));

    // cast shadow ray
    if ( nDl > 0.0f && LnDl > 0.0f ) {
      PerRayData_pathtrace_shadow shadow_prd;
      shadow_prd.inShadow = false;
      Ray shadow_ray = make_Ray( hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist );
      rtTrace(top_object, shadow_ray, shadow_prd);

      if(!shadow_prd.inShadow){
        float weight=nDl * LnDl * A / (M_PIf*Ldist*Ldist);
        result += light.emission * weight;
      }
    }
  }

  current_prd.radiance = result;
}

rtDeclareVariable(float3,        glass_color, , );
rtDeclareVariable(float,         index_of_refraction, , );

RT_PROGRAM void glass_refract()
{
  float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

  float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

  float3 hitpoint = ray.origin + t_hit * ray.direction;
  current_prd.origin = hitpoint;
  current_prd.countEmitted = true;
  float iof;
  if (current_prd.inside) {
    // Shoot outgoing ray
    iof = 1.0f/index_of_refraction;
  } else {
    iof = index_of_refraction;
  }
  refract(current_prd.direction, ray.direction, ffnormal, iof);
  //prd.direction = reflect(ray.direction, ffnormal);

  if (current_prd.inside) {
    // Compute Beer's law
    current_prd.attenuation = current_prd.attenuation * powf(glass_color, t_hit);
  }
  current_prd.inside = !current_prd.inside;

  current_prd.radiance = make_float3(0.0f);
}

//-----------------------------------------------------------------------------
//
//  Exception program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void exception()
{
  output_buffer[launch_index] = make_float4(bad_color, 0.0f);
}


//-----------------------------------------------------------------------------
//
//  Miss program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void miss()
{
  current_prd.radiance = bg_color;
  current_prd.done = true;
}


rtDeclareVariable(PerRayData_pathtrace_shadow, current_prd_shadow, rtPayload, );

RT_PROGRAM void shadow()
{
  current_prd_shadow.inShadow = true;
  rtTerminateRay();
}
